#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "index_loader.h"

#define POSTINGS_FILE "resources/seq_posting.txt"
#define POSTINGS_FILE2 "resources/mini_postings.txt"
#define POSTINGS_FILE3 "resources/mini_postings2.txt"
#define POSTINGS_FILE4 "resources/mini_seq_posting.txt"
#define DOCUMENTS_NORM "resources/documents_norm.txt"

Posting* postingsFromSeqFile(FILE *postingsFile, int totalTerms);
float* docsNormFromSeqFile(FILE *docsNormFile, int totalDocs);
void index_collection();
void resolveQuery(char *query);
void handleKernelError();
hipError_t checkCuda(hipError_t result);
// to use only during developing, delete on production
Posting* LoadDummyPostings(int size);
void displayPosting(Posting *postings, int size);

// global variables that are allocated in device during indexing
Posting *d_postings;
float *dev_docsNorm;
int terms;
int docs;

// GPU KERNEL
__global__ void k_resolveQuery (
		Posting *postings,
		float *docsNorm,
		int terms,
		int docs,
		int *queryTerms,
		int querySize,
		float *docScores
	){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= docs) return;

	int myDocId = index;
	printf("doc %d has norm %4.4f\n", myDocId, docsNorm[myDocId]);
	docScores[myDocId] = 0;
	int i, j, termId, termFound;
	for (i = 0; i < querySize; i++) {
		termId = queryTerms[i];
		termFound = j = 0;
		Posting termPosting;
		while (termFound != 1 && j < terms) {
			termPosting = postings[j];
			if (termPosting.termId == termId) termFound = 1;
			j++;
		}
		if (termFound == 1) {
			//printf("term %d has %d docs.\n", termPosting.termId, termPosting.docsLength);
			int docIdsPos = -1;
			int currentDocId;
			do {
				docIdsPos++;
				currentDocId = termPosting.docIds[docIdsPos];
				//printf("current doc id: %d\n", currentDocId);
			} while(currentDocId < myDocId && docIdsPos < termPosting.docsLength - 1);
			if (myDocId == currentDocId) {
				//printf("found my doc id: %d\n", currentDocId);
				//printf("doc %d: weight to sum: %4.2f\n", myDocId, termPosting.weights[docIdsPos]);
				docScores[myDocId] += termPosting.weights[docIdsPos];
				//printf("doc %d: current weight: %4.2f\n", myDocId, docScores[myDocId]);
			}
		}
	}
}

int main(int argc, char const *argv[]) {
  index_collection();

	/*
  char query[1000];
  printf("Enter query: ");
  fgets(query, 1000, stdin);
  if ((strlen(query)>0) && (query[strlen (query) - 1] == '\n'))
        query[strlen (query) - 1] = '\0';
  resolveQuery(query);
 	*/
	char src[3], query[3];

	strcpy(src,  "1 ");
	strcpy(query, "2 ");

	strcat(query, src);
	resolveQuery(query);
  return 0;
}


void index_collection() {

	terms = 346; // hardcoded
	docs = 6; // hardcoded

	printf("Loading postings...\n");
	FILE *txtFilePtr = fopen(POSTINGS_FILE4, "r");
	if(txtFilePtr == NULL) {
	 printf("Error! No posting file in path %s\n", POSTINGS_FILE4);
	 exit(1);
	}
  Posting* postingsLoaded = postingsFromSeqFile(txtFilePtr, terms);
  printf("Finish reading postings\n");

	// Postings to device
	printf("Copying postings from host to device\n");

  // POSTINGS TO DEVICE
	checkCuda( hipMalloc((void**)&d_postings, sizeof(Posting) * terms) );
	checkCuda( hipMemcpy(d_postings, postingsLoaded, sizeof(Posting) * terms, hipMemcpyHostToDevice) );
	int i;
	int *d_docIds;
	float *d_weights;
	for (i = 0; i < terms; i++) {
		Posting p = postingsLoaded[i];

		checkCuda( hipMalloc((void**) &d_docIds, sizeof(int) * p.docsLength) );
		checkCuda( hipMalloc((void**) &d_weights, sizeof(float) * p.docsLength) );

		checkCuda( hipMemcpy(&(d_postings[i].docIds), &(d_docIds), sizeof(int *), hipMemcpyHostToDevice) );
		checkCuda( hipMemcpy(&(d_postings[i].weights), &(d_weights), sizeof(float *), hipMemcpyHostToDevice) );

		checkCuda( hipMemcpy(d_docIds, p.docIds, sizeof(int) * p.docsLength, hipMemcpyHostToDevice) );
		checkCuda( hipMemcpy(d_weights, p.weights, sizeof(float) * p.docsLength, hipMemcpyHostToDevice) );

	}


	printf("Loading documents norm...\n");
	txtFilePtr = fopen(DOCUMENTS_NORM, "r");
	if(txtFilePtr == NULL) {
	 printf("Error! No documents norm file in path %s\n", DOCUMENTS_NORM);
	 exit(1);
	}
	float* documentsNorm = docsNormFromSeqFile(txtFilePtr, docs);
	printf("Finish loading documents norms\n");

	// docs norm to device
	printf("Copying docs norm from host to device\n");
	checkCuda( hipMalloc((void**)& dev_docsNorm, sizeof(float) * docs) );
	checkCuda( hipMemcpy(dev_docsNorm, documentsNorm, sizeof(float) * docs, hipMemcpyHostToDevice) );

	free(postingsLoaded);
	free(documentsNorm);
  printf("Finish indexing\n");
}

void resolveQuery(char *query){
  printf("Searching for: %s\n", query);
	hipEvent_t resolveQueryStart, resolveQueryStop;
	hipEventCreate(&resolveQueryStart);
	hipEventCreate(&resolveQueryStop);
  int i;
  int previousCharIsSpace = 0;
  int spacesCounter = 0;

  for (i = 0; i < strlen(query); i++) {
    if (query[i] != ' ') {
      previousCharIsSpace = 0;
    } else if (previousCharIsSpace == 0) {
      previousCharIsSpace = 1;
      spacesCounter++;
    }
  }
  int querySize = spacesCounter + 1;
  int *queryTerms = (int *) malloc(sizeof(int) * querySize);
  char *tokens = strtok(query, " ");
  int termPos = 0;
  while (tokens != NULL) {
    char *ptr;
    queryTerms[termPos] = strtol(tokens, &ptr, 10);
    tokens = strtok(NULL, " ");
    termPos++;
  }

  int *d_queryTerms;
	float *docScores, *d_docScores;
	int BLOCK_SIZE = 1024;
	int numBlocks = (docs + BLOCK_SIZE - 1) / BLOCK_SIZE;

	docScores = (float *) malloc(sizeof(float) * docs);
	hipMalloc((void **) &d_queryTerms, querySize * sizeof(int));
  hipMalloc((void **) &d_docScores, docs * sizeof(float));

  hipMemcpy(d_queryTerms, queryTerms, querySize * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(resolveQueryStart);
	k_resolveQuery<<<numBlocks, BLOCK_SIZE>>>(
		d_postings,
		dev_docsNorm,
		terms,
		docs,
		d_queryTerms,
		querySize,
		d_docScores
	);
	handleKernelError();
	hipEventRecord(resolveQueryStop);

	hipMemcpy(docScores, d_docScores, docs * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(resolveQueryStop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, resolveQueryStart, resolveQueryStop);

	printf("Time elapsed: %10.4f ms\n", milliseconds);

	for (i=0; i < docs; i++){
		printf("doc %d: %4.2f\n", i, docScores[i]);
	}

	hipFree(d_queryTerms);
	hipFree(d_docScores);
  free(queryTerms);
}

void handleKernelError(){
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
	  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
	  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n",
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}
