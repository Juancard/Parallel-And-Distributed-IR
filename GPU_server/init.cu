#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "index_loader.h"

#define POSTINGS_FILE "resources/seq_posting.txt"

typedef struct Lala {
   int termId;
	 int docsLength;
   int *docIds;
 } Lala;
// global variables that are allocated in device during indexing
Posting *d_postings;
int *d_termsInPostings;
Lala *d_lala;

// GPU KERNEL
__global__ void k_resolveQuery (
		Lala *lala,
		Posting *postings,
		int *termsInPostings,
		int *queryTerms,
		int querySize,
		float *docScores
	){
	printf("lala: %d\n", lala->docsLength);
	int myDocId = blockIdx.x * blockDim.x + threadIdx.x;
	docScores[myDocId] = 0;
	int i, j, termId, termFound;
	for (i = 0; i < querySize; i++) {
		termId = queryTerms[i];
		termFound = j = 0;
		Posting termPosting;
		while (termFound != 1 && j < *termsInPostings) {
			termPosting = postings[j];
			if (termPosting.termId == termId) termFound = 1;
			j++;
		}
		if (termFound == 1) {
			int docIdsPos = -1;
			int currentDocId;
			printf("%d\n", termId);
			do {
				docIdsPos++;
				printf("in do while %d\n", docIdsPos);
				currentDocId = termPosting.docIds[docIdsPos];
				printf("currentDocId: %d\n", currentDocId);
			} while(myDocId < currentDocId && docIdsPos < termPosting.docsLength - 1);
			if (myDocId == currentDocId) {
				docScores[myDocId] += termPosting.weights[docIdsPos];
			}
		}
	}
	/*
	docScores[index] = index + 0.0f;
	printf ("I am (%d, %d) with doc %d - score %1.1f\n", blockIdx.x, threadIdx.x, index, docScores[index]);
	printf("\t Term %d is in %d docs\n", index, postings[index].docsLength);
	*/
}

void displayPosting(Posting *postings, int size);
Posting* postingsFromSeqFile(FILE *postingsFile, int totalTerms);
void index_collection();
void resolveQuery(char *query);
void handleKernelError();


int main(int argc, char const *argv[]) {
  index_collection();

	/*
  char query[1000];
  printf("Enter query: ");
  fgets(query, 1000, stdin);
  if ((strlen(query)>0) && (query[strlen (query) - 1] == '\n'))
        query[strlen (query) - 1] = '\0';
  resolveQuery(query);
 	*/
	char src[3], query[3];

	strcpy(src,  "1 ");
	strcpy(query, "2 ");

	strcat(query, src);
	resolveQuery(query);
  return 0;
}


void index_collection() {
  FILE *txtFilePtr = fopen(POSTINGS_FILE, "r");
  if(txtFilePtr == NULL) {
   printf("Error! No posting file in path %s.\n", POSTINGS_FILE);
   exit(1);
  }
  const int TERMS = 30332;
	printf("Loading postings...\n");
  Posting* postingsLoaded = postingsFromSeqFile(txtFilePtr, TERMS);

	// Postings to device
	/*
	TODO structs ARE NOT COPIED THIS WAY, FIX IT ON NEXT COMMIT
	*/
	printf("Copying postings from host to device\n");
	int postingsSize = sizeof(Posting) * TERMS;
	hipMalloc((void **) &d_postings, postingsSize);
	hipMemcpy(d_postings, postingsLoaded, postingsSize, hipMemcpyHostToDevice);

	// terms to device
	hipMalloc((void **) &d_termsInPostings, sizeof(int));
  hipMemcpy(d_termsInPostings, &TERMS, sizeof(int), hipMemcpyHostToDevice);

	free(postingsLoaded);
  printf("Finish indexing\n");
}

void resolveQuery(char *query){
  printf("Searching for: %s\n", query);
	hipEvent_t resolveQueryStart, resolveQueryStop;
	hipEventCreate(&resolveQueryStart);
	hipEventCreate(&resolveQueryStop);
  int i;
  int previousCharIsSpace = 0;
  int spacesCounter = 0;

  for (i = 0; i < strlen(query); i++) {
    if (query[i] != ' ') {
      previousCharIsSpace = 0;
    } else if (previousCharIsSpace == 0) {
      previousCharIsSpace = 1;
      spacesCounter++;
    }
  }
  int querySize = spacesCounter + 1;
  int *queryTerms = (int *) malloc(sizeof(int) * querySize);
  char *tokens = strtok(query, " ");
  int termPos = 0;
  while (tokens != NULL) {
    char *ptr;
    queryTerms[termPos] = strtol(tokens, &ptr, 10);
    tokens = strtok(NULL, " ");
    termPos++;
  }

  int *d_queryTerms;
	float *docScores, *d_docScores;
	int DOCS = 4;
	int BLOCK_SIZE = DOCS;//1024;

	int numBlocks = (DOCS + BLOCK_SIZE - 1) / BLOCK_SIZE;

	docScores = (float *) malloc(sizeof(float) * DOCS);
	hipMalloc((void **) &d_queryTerms, querySize * sizeof(int));
  hipMalloc((void **) &d_docScores, DOCS * sizeof(float));

  hipMemcpy(d_queryTerms, queryTerms, querySize * sizeof(int), hipMemcpyHostToDevice);

	// Allocate storage for struct LALA and docIds
	Lala l;
	l.termId = 5;
	l.docsLength = 1;
	int size =  sizeof(int) * l.docsLength;
	l.docIds = (int *) malloc(size);
	int *d_docIds;
  hipMalloc(&d_lala, sizeof(Lala));
  hipMalloc(&d_docIds, size);

	hipMemcpy(d_docIds, l.docIds, size, hipMemcpyHostToDevice);
	hipMemcpy(d_lala, &l, sizeof(Lala), hipMemcpyHostToDevice);
	hipMemcpy(&(d_lala->docIds), &d_lala, sizeof(int*), hipMemcpyHostToDevice);

	hipEventRecord(resolveQueryStart);
	k_resolveQuery<<<numBlocks, BLOCK_SIZE>>>(
		d_lala,
		d_postings,
		d_termsInPostings,
		d_queryTerms,
		querySize,
		d_docScores
	);
	handleKernelError();
	hipEventRecord(resolveQueryStop);

	hipMemcpy(docScores, d_docScores, DOCS * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(resolveQueryStop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, resolveQueryStart, resolveQueryStop);

	printf("Time elapsed: %10.4f ms\n", milliseconds);

	hipFree(d_queryTerms);
	hipFree(d_docScores);
  free(queryTerms);
}

void handleKernelError(){
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
	  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
	  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}
