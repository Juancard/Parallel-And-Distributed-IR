#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "index_loader.h"

#define POSTINGS_FILE "resources/seq_posting.txt"
#define POSTINGS_FILE2 "resources/mini_postings.txt"
#define POSTINGS_FILE3 "resources/mini_postings2.txt"
#define POSTINGS_FILE4 "resources/mini_seq_posting.txt"
#define DOCUMENTS_NORM "resources/documents_norm.txt"

typedef struct Query {
   int size;
   float *weights;
   int *termsId;
	 float norm;
 } Query;

Posting* postingsFromSeqFile(FILE *postingsFile, int totalTerms);
float* docsNormFromSeqFile(FILE *docsNormFile, int totalDocs);
void index_collection();
void resolveQuery(char *query);
Query parseQuery(char* queryStr);
void handleKernelError();
hipError_t checkCuda(hipError_t result);
// to use only during developing, delete on production
Posting* LoadDummyPostings(int size);
void displayPosting(Posting *postings, int size);



// global variables that are allocated in device during indexing
Posting *dev_postings;
float *dev_docsNorm;
int terms;
int docs;

// GPU KERNEL
__global__ void k_resolveQuery (
		Posting *postings,
		float *docsNorm,
		int terms,
		int docs,
		Query q,
		float *docScores
	){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= docs) return;

	int myDocId = index;
	//printf("doc %d has norm %4.4f\n", myDocId, docsNorm[myDocId]);
	docScores[myDocId] = 0;
	int i;
	Posting termPosting;
	//printf("query terms: %d\n", querySize);
	for (i = 0; i < q.size; i++) {
		termPosting = postings[q.termsId[i]];
		//printf("term %d has %d docs.\n", q.termsId[i], termPosting.docsLength);
		int docIdsPos = -1;
		int currentDocId;
		do {
			docIdsPos++;
			currentDocId = termPosting.docIds[docIdsPos];
			//printf("current doc id: %d\n", currentDocId);
		} while(currentDocId < myDocId && docIdsPos < termPosting.docsLength - 1);
		if (myDocId == currentDocId) {
			//printf("found my doc id: %d\n", currentDocId);
			//printf("doc %d: weight to sum: %4.2f\n", myDocId, termPosting.weights[docIdsPos]);
			docScores[myDocId] += termPosting.weights[docIdsPos];
			//printf("doc %d: current weight: %4.2f\n", myDocId, docScores[myDocId]);
		}
	}
}

int main(int argc, char const *argv[]) {
  index_collection();

	/*
  char query[1000];
  printf("Enter query: ");
  fgets(query, 1000, stdin);
  if ((strlen(query)>0) && (query[strlen (query) - 1] == '\n'))
        query[strlen (query) - 1] = '\0';
  resolveQuery(query);
 	*/
	char q[20];

	/* Query string format:
	[norma_query]#[term_1]:[weight_1];[term_n]:[weight_n]
	*/
	strcpy(q, "1.4142135624#10:1;11:1;");

	resolveQuery(q);
  return 0;
}


void index_collection() {

	terms = 17; // hardcoded
	docs = 4; // hardcoded

	printf("Loading postings...\n");
	FILE *txtFilePtr = fopen(POSTINGS_FILE3, "r");
	if(txtFilePtr == NULL) {
	 printf("Error! No posting file in path %s\n", POSTINGS_FILE3);
	 exit(1);
	}
  Posting* postingsLoaded = postingsFromSeqFile(txtFilePtr, terms);
  printf("Finish reading postings\n");

	// Postings to device
	printf("Copying postings from host to device\n");

  // POSTINGS TO DEVICE
	checkCuda( hipMalloc((void**)&dev_postings, sizeof(Posting) * terms) );
	checkCuda( hipMemcpy(dev_postings, postingsLoaded, sizeof(Posting) * terms, hipMemcpyHostToDevice) );
	int i;
	int *dev_docIds;
	float *dev_weights;
	for (i = 0; i < terms; i++) {
		Posting p = postingsLoaded[i];

		checkCuda( hipMalloc((void**) &dev_docIds, sizeof(int) * p.docsLength) );
		checkCuda( hipMalloc((void**) &dev_weights, sizeof(float) * p.docsLength) );

		checkCuda( hipMemcpy(&(dev_postings[i].docIds), &(dev_docIds), sizeof(int *), hipMemcpyHostToDevice) );
		checkCuda( hipMemcpy(&(dev_postings[i].weights), &(dev_weights), sizeof(float *), hipMemcpyHostToDevice) );

		checkCuda( hipMemcpy(dev_docIds, p.docIds, sizeof(int) * p.docsLength, hipMemcpyHostToDevice) );
		checkCuda( hipMemcpy(dev_weights, p.weights, sizeof(float) * p.docsLength, hipMemcpyHostToDevice) );

	}


	printf("Loading documents norm...\n");
	txtFilePtr = fopen(DOCUMENTS_NORM, "r");
	if(txtFilePtr == NULL) {
	 printf("Error! No documents norm file in path %s\n", DOCUMENTS_NORM);
	 exit(1);
	}
	float* documentsNorm = docsNormFromSeqFile(txtFilePtr, docs);
	printf("Finish loading documents norms\n");

	// docs norm to device
	printf("Copying docs norm from host to device\n");
	checkCuda( hipMalloc((void**)& dev_docsNorm, sizeof(float) * docs) );
	checkCuda( hipMemcpy(dev_docsNorm, documentsNorm, sizeof(float) * docs, hipMemcpyHostToDevice) );

	free(postingsLoaded);
	free(documentsNorm);
  printf("Finish indexing\n");
}

void resolveQuery(char *queryStr){
  printf("Searching for: %s\n", queryStr);
	hipEvent_t resolveQueryStart, resolveQueryStop;
	hipEventCreate(&resolveQueryStart);
	hipEventCreate(&resolveQueryStop);
  int i;
	Query q = parseQuery(queryStr);
	printf("Query: \n");
	for (i = 0; i < q.size; i++) {
		printf("term %d: %.4f\n", q.termsId[i], q.weights[i]);
	}
	printf("Query norm: %.4f\n", q.norm);
	exit(0);
	/*
  int previousCharIsSpace = 0;
  int spacesCounter = 0;

  for (i = 0; i < strlen(queryStr); i++) {
    if (query[i] != ' ') {
      previousCharIsSpace = 0;
    } else if (previousCharIsSpace == 0) {
      previousCharIsSpace = 1;
      spacesCounter++;
    }
  }
  int querySize = spacesCounter + 1;

  int *queryTerms = (int *) malloc(sizeof(int) * q.size);
  char *tokens = strtok(query, " ");
  int termPos = 0;
  while (tokens != NULL) {
    char *ptr;
    queryTerms[termPos] = strtol(tokens, &ptr, 10);
    tokens = strtok(NULL, " ");
    termPos++;
  }

  int *dev_queryTerms;
		*/

	float *docScores, *dev_docScores;
	int BLOCK_SIZE = 1024;
	int numBlocks = (docs + BLOCK_SIZE - 1) / BLOCK_SIZE;

	docScores = (float *) malloc(sizeof(float) * docs);
	//hipMalloc((void **) &dev_queryTerms, querySize * sizeof(int));
  hipMalloc((void **) &dev_docScores, docs * sizeof(float));

  //hipMemcpy(dev_queryTerms, queryTerms, querySize * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(resolveQueryStart);
	k_resolveQuery<<<numBlocks, BLOCK_SIZE>>>(
		dev_postings,
		dev_docsNorm,
		terms,
		docs,
		q,
		dev_docScores
	);
	handleKernelError();
	hipEventRecord(resolveQueryStop);

	hipMemcpy(docScores, dev_docScores, docs * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(resolveQueryStop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, resolveQueryStart, resolveQueryStop);

	printf("Time elapsed: %10.4f ms\n", milliseconds);

	for (i=0; i < docs; i++){
		printf("doc %d: %4.2f\n", i, docScores[i]);
	}

	hipFree(dev_docScores);
	/*
	TODO: FREE QUERY
	*/
}

Query parseQuery(char* queryStr){
	Query q;
	char *tokens = strtok(queryStr, "#");
	q.norm = atof(tokens);
	char *termToWeight = strtok(NULL, "#");
	q.size = 0;
	int i;
	for (i=0; i < strlen(termToWeight); i++)
		if (termToWeight[i] == ';')
			q.size++;
	//printf("terms length: %d\n", q.size);
	q.termsId = (int*) malloc(sizeof(int) * q.size);
	q.weights = (float*) malloc(sizeof(float) * q.size);
	char *tokenPtr1, *termStr, *weightStr;
	tokens = strtok_r(termToWeight, ";", &tokenPtr1);
	int termPos = 0;
	while (tokens != NULL) {
		char *tokenPtr2, *intPtr;
		termStr = strtok_r(tokens, ":", &tokenPtr2);
		weightStr = strtok_r(NULL, ":", &tokenPtr2);
		q.weights[termPos] = atof(weightStr);
		q.termsId[termPos] = strtol(termStr, &intPtr, 10);
		tokens = strtok_r(NULL, ";", &tokenPtr1);
		termPos++;
	}
	return q;
}

void handleKernelError(){
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
	  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
	  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n",
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}
